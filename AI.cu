#include <iostream>

#include "AI.h"

bool checkForError(hipError_t error)
{
    if (error != hipSuccess) {
        std::cout << hipGetErrorString(error) << std::endl;
    }

    return error != hipSuccess;
}

bool checkForKernelError(const char *errorMessage)
{
    hipError_t status = hipGetLastError();
    if (status != hipSuccess) {
        std::cout << errorMessage << hipGetErrorString(status) << std::endl;
    }

    return status != hipSuccess;
}
